#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
// upfirRow kernel
// Upsampling by 2 and filtering along 1st dimension
////////////////////////////////////////////////////////////////////////////////
__global__ void upfirRow(
    double *d_Dst,
    double *d_Src,
    double *d_Kernel,
    int inVolRowSize,
    int inVolColSize,
    int inVolBeaSize,
	int kernelLength
){
	int outVolRowSize = int(inVolRowSize * 2 + kernelLength - 2);
	int outVolColSize = inVolColSize;
	int outVolBeaSize = inVolBeaSize;

//	int inI = blockIdx.x * blockDim.x + threadIdx.x;
//	int inJ = blockIdx.y * blockDim.y + threadIdx.y;
//	int inK = blockIdx.z * blockDim.z + threadIdx.z;
//	int outJ = inJ;
//	int outK = inK;

	int outI = blockIdx.x * blockDim.x + threadIdx.x;
	int outJ = blockIdx.y * blockDim.y + threadIdx.y;
	int outK = blockIdx.z * blockDim.z + threadIdx.z;

	int inI = int(0.0);
	int inJ = outJ;
	int inK = outK;
	int upperBound = 0;

	double sum = 0.0;

	if((outI < outVolRowSize) && (outJ < outVolColSize) && (outK < outVolBeaSize)) {
		if (outI % 2 == 0) {
			inI = outI / 2;
			if (kernelLength % 2 == 0)
				upperBound = kernelLength / 2;
			else
				upperBound = (kernelLength + 1) / 2;

			#pragma unroll
			for (int m = 0; m < upperBound; m++) {
				if ((inI - m) >= 0 && (inI - m) < inVolRowSize)
					sum += d_Kernel[m * 2] * d_Src[(inI - m) + inJ*inVolRowSize + inK*inVolRowSize*inVolColSize];
				else
					sum += 0.0;
			}
		}
		else {
			inI = (outI - 1) / 2;
			if (kernelLength % 2 == 0)
				upperBound = kernelLength / 2;
			else
				upperBound = (kernelLength - 1) / 2;

			#pragma unroll
			for (int m = 0; m < upperBound; m++) {
				if ((inI - m) >= 0 && (inI - m) < inVolRowSize)
					sum += d_Kernel[m * 2 + 1] * d_Src[(inI - m) + inJ*inVolRowSize + inK*inVolRowSize*inVolColSize];
				else
					sum += 0.0;
			}
		}	
		d_Dst[outI + outJ*outVolRowSize + outK*outVolRowSize*outVolColSize] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////
// upfirCol kernel
// Upsampling by 2 and filtering along 2nd dimension
////////////////////////////////////////////////////////////////////////////////
__global__ void upfirCol(
    double *d_Dst,
    double *d_Src,
    double *d_Kernel,
    int inVolRowSize,
    int inVolColSize,
    int inVolBeaSize,
	int kernelLength
){
	int outVolRowSize = inVolRowSize;
	int outVolColSize = int(inVolColSize * 2 + kernelLength - 2);
	int outVolBeaSize = inVolBeaSize;

	int outI = blockIdx.x * blockDim.x + threadIdx.x;
	int outJ = blockIdx.y * blockDim.y + threadIdx.y;
	int outK = blockIdx.z * blockDim.z + threadIdx.z;

	int inI = outI;
	int inJ = int(0.0);
	int inK = outK;
	int upperBound = 0;

	double sum = 0.0;

	if((outI < outVolRowSize) && (outJ < outVolColSize) && (outK < outVolBeaSize)) {
		if (outJ % 2 == 0) {
			inJ = outJ / 2;
			if (kernelLength % 2 == 0)
				upperBound = kernelLength / 2;
			else
				upperBound = (kernelLength + 1) / 2;

			#pragma unroll
			for (int m = 0; m < upperBound; m++) {
				if ((inJ - m) >= 0 && (inJ - m) < inVolColSize)
					sum += d_Kernel[m * 2] * d_Src[inI + (inJ - m)*inVolRowSize + inK*inVolRowSize*inVolColSize];
				else
					sum += 0.0;
			}
		}
		else {
			inJ = (outJ - 1) / 2;
			if (kernelLength % 2 == 0)
				upperBound = kernelLength / 2;
			else
				upperBound = (kernelLength - 1) / 2;

			#pragma unroll
			for (int m = 0; m < upperBound; m++) {
				if ((inJ - m) >= 0 && (inJ - m) < inVolColSize)
					sum += d_Kernel[m * 2 + 1] * d_Src[inI + (inJ - m)*inVolRowSize + inK*inVolRowSize* ];
				else
					sum += 0.0;
			}
		}	
		d_Dst[outI + outJ*outVolRowSize + outK*outVolRowSize*outVolColSize] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////
// upfirBea kernel
// Upsampling by 2 and filtering along 3rd dimension
////////////////////////////////////////////////////////////////////////////////
__global__ void upfirBea(
    double *d_Dst,
    double *d_Src,
    double *d_Kernel,
    int inVolRowSize,
    int inVolColSize,
    int inVolBeaSize,
	int kernelLength
){
	int outVolRowSize = inVolRowSize;
	int outVolColSize = inVolColSize;
	int outVolBeaSize = int(inVolBeaSize * 2 + kernelLength - 2);

	int outI = blockIdx.x * blockDim.x + threadIdx.x;
	int outJ = blockIdx.y * blockDim.y + threadIdx.y;
	int outK = blockIdx.z * blockDim.z + threadIdx.z;

	int inI = outI;
	int inJ = outJ;
	int inK = int(0.0);
	int upperBound = 0;

	double sum = 0.0;

	if((outI < outVolRowSize) && (outJ < outVolColSize) && (outK < outVolBeaSize)) {
		if (outK % 2 == 0) {
			inK = outK / 2;
			if (kernelLength % 2 == 0)
				upperBound = kernelLength / 2;
			else
				upperBound = (kernelLength + 1) / 2;

			#pragma unroll
			for (int m = 0; m < upperBound; m++) {
				if ((inK - m) >= 0 && (inK - m) < inVolBeaSize)
					sum += d_Kernel[m * 2] * d_Src[inI + inJ*inVolRowSize + (inK - m)*inVolRowSize*inVolColSize];
				else
					sum += 0.0;
			}
		}
		else {
			inK = (outK - 1) / 2;
			if (kernelLength % 2 == 0)
				upperBound = kernelLength / 2;
			else
				upperBound = (kernelLength - 1) / 2;

			#pragma unroll
			for (int m = 0; m < upperBound; m++) {
				if ((inK - m) >= 0 && (inK - m) < inVolBeaSize)
					sum += d_Kernel[m * 2 + 1] * d_Src[inI + inJ*inVolRowSize + (inK - m)*inVolRowSize*inVolColSize];
				else
					sum += 0.0;
			}
		}	
		d_Dst[outI + outJ*outVolRowSize + outK*outVolRowSize*outVolColSize] = sum;
	}
}