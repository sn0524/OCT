#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
// firDnRow kernel
// filtering and downsampling by 2 along 1st dimension
////////////////////////////////////////////////////////////////////////////////
__global__ void firDnRow(
    double *d_Dst,
    double *d_Src,
    double *d_Kernel,
    int inVolRowSize,
    int inVolColSize,
    int inVolBeaSize,
	int kernelLength
){
	int convLength = int(inVolRowSize + kernelLength - 1);
	int outVolRowSize = int(0.0);
	int outVolColSize = inVolColSize;
	int outVolBeaSize = inVolBeaSize;

	if (convLength % 2 == 0)
		outVolRowSize = convLength / 2;	
	else
		outVolRowSize = (convLength + 1) / 2;

	int outI = blockIdx.x * blockDim.x + threadIdx.x;
	int outJ = blockIdx.y * blockDim.y + threadIdx.y;
	int outK = blockIdx.z * blockDim.z + threadIdx.z;

	int inI = int(outI * 2);
	int inJ = outJ;
	int inK = outK;

	double sum = 0.0;

//    int lowerBound = 0;
//   if (kernelRadius % 2 == 0)
//        lowerBound = -kernelRadius+1;
//    else
//        lowerBound = -kernelRadius;

	if((outI < outVolRowSize) && (outJ < outVolColSize) && (outK < outVolBeaSize)) {
//		#pragma unroll
//		for(int m = lowerBound; m <= kernelRadius; m++)
//		{
//			if ( (inI+m) >= 0 && (inI+m) < inVolRowSize )
//				sum += d_Kernel[kernelRadius - m] * d_Src[(inI+m) + inJ*inVolRowSize + inK*inVolRowSize*inVolColSize];
//			else
//				sum += d_Kernel[kernelRadius - m] * 0.0;
//		}
//		d_Dst[outI + outJ*outVolRowSize + outK*outVolRowSize*outVolColSize] = sum;
	
		#pragma unroll
		for (int m = 0; m < kernelLength; m++) {
			if ((inI - m) >= 0 && (inI - m) < inVolRowSize)
				sum += d_Kernel[m] * d_Src[(inI - m) + inJ*inVolRowSize + inK*inVolRowSize*inVolColSize];
			else
				sum += 0.0;
		}
		d_Dst[outI + outJ*outVolRowSize + outK*outVolRowSize*outVolColSize] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////
// firDnCol kernel
// filtering and downsampling by 2 along 2nd dimension
////////////////////////////////////////////////////////////////////////////////
__global__ void firDnCol(
    double *d_Dst,
    double *d_Src,
    double *d_Kernel,
    int inVolRowSize,
    int inVolColSize,
    int inVolBeaSize,
	int kernelLength
){
	int convLength = int(inVolColSize + kernelLength - 1);
	int outVolRowSize = inVolRowSize;
	int outVolColSize = int(0.0);
	int outVolBeaSize = inVolBeaSize;

	if (convLength % 2 == 0)
		outVolColSize = convLength / 2;	
	else
		outVolColSize = (convLength + 1) / 2;

	int outI = blockIdx.x * blockDim.x + threadIdx.x;
	int outJ = blockIdx.y * blockDim.y + threadIdx.y;
	int outK = blockIdx.z * blockDim.z + threadIdx.z;

	int inI = outI;
	int inJ = int(outJ * 2);
	int inK = outK;

	double sum = 0.0;

	if((outI < outVolRowSize) && (outJ < outVolColSize) && (outK < outVolBeaSize)) {
		#pragma unroll
		for (int m = 0; m < kernelLength; m++) {
			if ((inJ - m) >= 0 && (inJ - m) < inVolColSize)
				sum += d_Kernel[m] * d_Src[inI + (inJ - m)*inVolRowSize + inK*inVolRowSize*inVolColSize];
			else
				sum += 0.0;
		}
		d_Dst[outI + outJ*outVolRowSize + outK*outVolRowSize*outVolColSize] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////
// firDnBea kernel
// filtering and downsampling by 2 along 3rd dimension
////////////////////////////////////////////////////////////////////////////////
__global__ void firDnBea(
    double *d_Dst,
    double *d_Src,
    double *d_Kernel,
    int inVolRowSize,
    int inVolColSize,
    int inVolBeaSize,
	int kernelLength
){
	int convLength = int(inVolBeaSize + kernelLength - 1);
	int outVolRowSize = inVolRowSize;
	int outVolColSize = inVolColSize;
	int outVolBeaSize = int(0.0);

	if (convLength % 2 == 0)
		outVolBeaSize = convLength / 2;	
	else
		outVolBeaSize = (convLength + 1) / 2;

	int outI = blockIdx.x * blockDim.x + threadIdx.x;
	int outJ = blockIdx.y * blockDim.y + threadIdx.y;
	int outK = blockIdx.z * blockDim.z + threadIdx.z;

	int inI = outI;
	int inJ = outJ;
	int inK = int(outK * 2);

	double sum = 0.0;

	if((outI < outVolRowSize) && (outJ < outVolColSize) && (outK < outVolBeaSize)) {
		#pragma unroll
		for (int m = 0; m < kernelLength; m++) {
			if ((inK - m) >= 0 && (inK - m) < inVolBeaSize)
				sum += d_Kernel[m] * d_Src[inI + inJ*inVolRowSize + (inK - m)*inVolRowSize*inVolColSize];
			else
				sum += 0.0;
		}
		d_Dst[outI + outJ*outVolRowSize + outK*outVolRowSize*outVolColSize] = sum;
	}
}